#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_specific_margin_layer.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void ArcCosDegree(const int n, const Dtype* in, Dtype* out) {
    CUDA_KERNEL_LOOP(index, n) {
      Dtype fixed_in_data = min(in[index], Dtype(1.0) - Dtype(1e-4));
      fixed_in_data = max(fixed_in_data, Dtype(-1.0) + Dtype(1e-4));
      out[index] = Dtype(acos(in[index]) / M_PI * 180.0);
    }
  }

  template <typename Dtype>
  __global__ void CreateMask(const int num, const int dim, const Dtype* label, Dtype* positive_mask, Dtype* negative_mask) {
    CUDA_KERNEL_LOOP(index, num) {
      int gt = static_cast<int>(label[index]);
      positive_mask[index*dim + gt] = Dtype(1);
      negative_mask[index*dim + gt] = Dtype(0);
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificSoftMarginForward(const int n, const int dim, const Dtype* bottom_data, const Dtype* label,
                                                 Dtype* top_data, Dtype* theta, Dtype margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      theta[index * dim + gt] = acos(bottom_data[index * dim + gt]);
      if (margin * theta[index * dim + gt] > M_PI - 1e-4) {
        theta[index * dim + gt] = M_PI - 1e-4;
      }
      top_data[index * dim + gt] = cos(margin * theta[index * dim + gt]);
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificSoftMarginBackward(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                  Dtype* bottom_diff, const Dtype* bottom_data, const Dtype* theta, const Dtype* top_data, Dtype margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      Dtype fixed_bottom_data = min(bottom_data[index * dim + gt], Dtype(1.0) - Dtype(1e-4));
      fixed_bottom_data = max(fixed_bottom_data, Dtype(-1.0) + Dtype(1e-4));
      Dtype gradient = margin * sin(margin * theta[index * dim + gt]) / sqrt(1 - fixed_bottom_data * fixed_bottom_data);
      gradient = gradient > 2 ? 2 : gradient;//bound the gradient.
      gradient = gradient < 0 ? 0 : gradient;
      bottom_diff[index * dim + gt] = top_diff[index * dim + gt] * gradient;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginForward(const int n, const int dim, const Dtype* bottom_data, const Dtype* label,
                                                 Dtype* top_data, Dtype cos_margin, Dtype sin_margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      Dtype fixed_bottom_data = min(bottom_data[index * dim + gt], Dtype(1.0) - Dtype(1e-4));
      fixed_bottom_data = max(fixed_bottom_data, Dtype(-1.0) + Dtype(1e-4));
      top_data[index * dim + gt] = fixed_bottom_data * cos_margin -
        sqrt(1 - fixed_bottom_data * fixed_bottom_data) * sin_margin;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginBackward(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                  Dtype* bottom_diff, const Dtype* bottom_data, Dtype cos_margin, Dtype sin_margin) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      Dtype fixed_bottom_data = min(bottom_data[index * dim + gt], Dtype(1.0) - Dtype(1e-4));
      fixed_bottom_data = max(fixed_bottom_data, Dtype(-1.0) + Dtype(1e-4));
      Dtype gradient = cos_margin + fixed_bottom_data / sqrt(1 - fixed_bottom_data * fixed_bottom_data) * sin_margin;
      gradient = gradient > 2 ? 2 : gradient;//bound the gradient.
      gradient = gradient < 0 ? 0 : gradient;
      bottom_diff[index * dim + gt] = top_diff[index * dim + gt] * gradient;
    }
  }

  template <typename Dtype>
  __global__ void LabelSpecificHardMarginBackwardToMargin(const int n, const int dim, const Dtype* top_diff, const Dtype* label,
                                                          Dtype* margin_diff, const Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      margin_diff[index] = top_diff[index * dim + gt] * sqrt(1 - top_data[index * dim + gt] * top_data[index * dim + gt]);
    }
  }

  template <typename Dtype>
  void LabelSpecificMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    Dtype* margin = this->blobs_[0]->mutable_cpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    if (has_margin_base_ && this->phase_ == TRAIN) {
      margin[0] = margin_base_ + pow(((Dtype)1. + gamma_ * iter_), power_) - 1;
      iter_++;
    }
    if (has_margin_max_ && this->phase_ == TRAIN) {
      margin[0] = std::min(margin[0], margin_max_);
    }

    if (top.size() == 2 && auto_tune_) {
      Dtype *positive_mask_data = positive_mask.mutable_gpu_data();
      Dtype *negative_mask_data = negative_mask.mutable_gpu_data();
      caffe_gpu_set(count, Dtype(0), positive_mask_data);
      caffe_gpu_set(count, Dtype(1), negative_mask_data);
      // NOLINT_NEXT_LINE(whitespace/operators)
      CreateMask<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, label_data, positive_mask.mutable_gpu_data(), negative_mask.mutable_gpu_data());
      CUDA_POST_KERNEL_CHECK;

      Dtype positive_mean;
      //Dtype positive_std;
      Dtype negative_mean;
      //Dtype negative_std;

      // NOLINT_NEXT_LINE(whitespace/operators)
      ArcCosDegree<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> > (
        count, bottom_data, bottom_angle.mutable_gpu_data());
      //caffe_gpu_powx(count, bottom_angle.gpu_data(), Dtype(2), bottom_square.mutable_gpu_data());
      caffe_gpu_dot(count, bottom_angle.gpu_data(), positive_mask.gpu_data(), &positive_mean);
      //caffe_gpu_dot(count, bottom_square.gpu_data(), positive_mask.gpu_data(), &positive_std);
      caffe_gpu_dot(count, bottom_angle.gpu_data(), negative_mask.gpu_data(), &negative_mean);
      //caffe_gpu_dot(count, bottom_square.gpu_data(), negative_mask.gpu_data(), &negative_std);

      positive_mean /= num;
      //positive_std = sqrt(positive_std / num - positive_mean * positive_mean);
      negative_mean /= num * (dim - 1);
      //negative_std = sqrt(negative_std / num / (dim - 1) - negative_mean * negative_mean);

      if (this->phase_ == TEST) {
        top[1]->mutable_cpu_data()[0] = margin[0];
        top[1]->mutable_cpu_data()[1] = positive_mean;
        //top[1]->mutable_cpu_data()[2] = positive_std;
        top[1]->mutable_cpu_data()[2] = negative_mean;
        //top[1]->mutable_cpu_data()[4] = negative_std;
      }
      else {
        if (iter_ == 1) {
          margin[1] = positive_mean;
          //margin[2] = positive_std;
          margin[2] = negative_mean;
          //margin[4] = negative_std;
        }
        else {
          margin[1] = 0.99 * margin[1] + 0.01 * positive_mean;
          //margin[2] = 0.99 * margin[2] + 0.01 * positive_std;
          margin[2] = 0.99 * margin[2] + 0.01 * negative_mean;
          //margin[4] = 0.99 * margin[4] + 0.01 * negative_std;
        }

        //margin[0] = (margin[3] - margin[1]) / (margin[2] + margin[4]) * margin[2];
        margin[0] = (margin[2] - margin[1]) / 2;
        caffe_copy(3, this->blobs_[0]->cpu_data(), top[1]->mutable_cpu_data());
      }
    }
    if (bottom.size() == 3) {
      margin[0] = bottom[2]->cpu_data()[0];
    }
    if (top.size() >= 2) {
      top[1]->mutable_cpu_data()[0] = margin[0];
    }

    caffe_copy(count, bottom_data, top_data);
    if (!margin_on_test_ && this->phase_ == TEST) return;

    if (margin[0] != Dtype(0.0)) {
      if (type_ == LabelSpecificMarginParameter_MarginType_SOFT) {
        // NOLINT_NEXT_LINE(whitespace/operators)
        LabelSpecificSoftMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
          num, dim, bottom_data, label_data, top_data, theta.mutable_gpu_data(), margin[0]);
        CUDA_POST_KERNEL_CHECK;
      }
      else {
        // NOLINT_NEXT_LINE(whitespace/operators)
        LabelSpecificHardMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
          num, dim, bottom_data, label_data, top_data, cos(margin[0] / 180 * M_PI), sin(margin[0] / 180 * M_PI));
        CUDA_POST_KERNEL_CHECK;
      }
    }
  }

  template <typename Dtype>
  void LabelSpecificMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    if (propagate_down[0]) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      const Dtype* label_data = bottom[1]->gpu_data();
      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* top_data = top[0]->gpu_data();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      Dtype* margin = this->blobs_[0]->mutable_cpu_data();

      int num = bottom[0]->num();
      int count = bottom[0]->count();
      int dim = count / num;

      caffe_copy(count, top_diff, bottom_diff);
      if (!margin_on_test_ && this->phase_ == TEST) return;
      if (pass_bp_) return;

      if (margin[0] != Dtype(0.0)) {
        if (type_ == LabelSpecificMarginParameter_MarginType_SOFT) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          LabelSpecificSoftMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
            num, dim, top_diff, label_data, bottom_diff, bottom_data, theta.gpu_data(), top_data, margin[0]);
          CUDA_POST_KERNEL_CHECK;
        }
        else {
          // NOLINT_NEXT_LINE(whitespace/operators)
          LabelSpecificHardMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
            num, dim, top_diff, label_data, bottom_diff, bottom_data, cos(margin[0] / 180 * M_PI), sin(margin[0] / 180 * M_PI));
          CUDA_POST_KERNEL_CHECK;
          if (bottom.size() == 3 && propagate_down[3]) {
            // NOLINT_NEXT_LINE(whitespace/operators)
            LabelSpecificHardMarginBackwardToMargin<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
              num, dim, top_diff, label_data, positive_data.mutable_gpu_data(), top_data);
            CUDA_POST_KERNEL_CHECK;
            caffe_gpu_dot(num, positive_data.gpu_data(), sum_multiplier_.gpu_data(), bottom[3]->mutable_cpu_data());
          }
        }
      }
    }
  }


  INSTANTIATE_LAYER_GPU_FUNCS(LabelSpecificMarginLayer);


}  // namespace caffe