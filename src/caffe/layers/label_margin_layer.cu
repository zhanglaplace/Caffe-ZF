#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_margin_layer.hpp"

namespace caffe {


	template <typename Dtype>
	__global__ void LabelMarginForward(const int n, const int dim, const Dtype* label,
		Dtype* top_data, const Dtype* bottom_data,Dtype cos_m,const Dtype* sqrt_sin_data,Dtype sin_m) {
		CUDA_KERNEL_LOOP(index, n) {
			int gt = static_cast<int>(label[index]);
			if (bottom_data[index * dim + gt] > sin_m) 
				top_data[index * dim + gt] = bottom_data[index* dim+ gt]*cos_m - sqrt(sqrt_sin_data[index*dim+gt])*sin_m;
		}
	}

	template <typename Dtype>
	void LabelMarginLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		Dtype cos_m = cos(bias_);
		Dtype sin_m = cos(bias_);

		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* label_data = bottom[1]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		int num = bottom[0]->num();
		int count = bottom[0]->count();
		int dim = count / num; // ͨ��Ϊ1

		caffe_copy(count, bottom_data, top_data);
		caffe_sqr(count, bottom_data, squar_sin_data.mutable_gpu_data());
		caffe_set(count,Dtype(1), one_data.mutable_gpu_data());
		caffe_cpu_axpby(count, Dtype(1), one_data.gpu_data(), Dtype(-1), squar_sin_data.mutable_gpu_data());

		if (!transform_test_ && this->phase_ == TEST) return;

		// NOLINT_NEXT_LINE(whitespace/operators)
		LabelMarginForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
			num, dim, label_data, top_data, bottom_data,cos_m,squar_sin_data.gpu_data(),sin_m);
		CUDA_POST_KERNEL_CHECK;
	}


	template <typename Dtype>
	__global__ void LabelMarginBackward(const int n, const int dim, const Dtype* label,
		Dtype* bottom_diff,  const Dtype* sqrt_sin_data, Dtype cos_m, const Dtype* bottom_data, Dtype sin_m) {
		CUDA_KERNEL_LOOP(index, n) {
			int gt = static_cast<int>(label[index]);
			if (bottom_data[index * dim + gt] > sin_m)
				bottom_diff[index * dim + gt] *= (-sqrt(sqrt_sin_data[index*dim+gt])*cos_m-bottom_data[index*dim+gt]*sin_m);
		}
	}


	template <typename Dtype>
	void LabelMarginLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		if (propagate_down[0]) {

			const Dtype* top_diff = top[0]->gpu_diff();
			const Dtype* bottom_data = bottom[0]->gpu_data();
            const Dtype* label_data = bottom[1]->gpu_data();
            Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();


			int count = top[0]->count();
			int num = bottom[0]->num();
			int dim = count / num; // ͨ��Ϊ1
			Dtype cos_m = cos(bias_);
			Dtype sin_m = cos(bias_);

			caffe_copy(count, top_diff, bottom_diff);

			if (!transform_test_ &&this->phase_ == TEST)
				return;

			LabelMarginBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
				num, dim, label_data, bottom_diff, squar_sin_data.gpu_data(), cos_m, bottom_data, sin_m);
			CUDA_POST_KERNEL_CHECK;

		}
	}
	INSTANTIATE_LAYER_GPU_FUNCS(LabelMarginLayer);
} // namespace caffe
